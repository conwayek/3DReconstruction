#include "hip/hip_runtime.h"
// ===============================================================================================================
// Copyright (c) 2019, Cornell University. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright otice, this list of conditions and
//       the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and
//       the following disclaimer in the documentation and/or other materials provided with the distribution.
//
//     * Neither the name of Cornell University nor the names of its contributors may be used to endorse or
//       promote products derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
// A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
// TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY
// OF SUCH DAMAGE.
//
// Author: Kai Zhang (kz298@cornell.edu)
//
// The research is based upon work supported by the Office of the Director of National Intelligence (ODNI),
// Intelligence Advanced Research Projects Activity (IARPA), via DOI/IBC Contract Number D17PC00287.
// The U.S. Government is authorized to reproduce and distribute copies of this work for Governmental purposes.
// ===============================================================================================================
//
//
// Copyright (c) 2022, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#define _USE_MATH_DEFINES

#include "mvs/patch_match_cuda.h"

#include <algorithm>
#include <cfloat>
#include <cmath>
#include <cstdint>
#include <sstream>
#include <cstdio>

#include "util/hip/hip_runtime.h"
#include "util/cudacc.h"
#include "util/logging.h"

// The number of threads per Cuda thread. Warning: Do not change this value,
// since the templated window sizes rely on this value.
#define THREADS_PER_BLOCK 32

// We must not include "util/math.h" to avoid any Eigen includes here,
// since Visual Studio cannot compile some of the Eigen/Boost expressions.
#ifndef DEG2RAD
#define DEG2RAD(deg) deg * 0.0174532925199432
#endif

namespace colmap {
namespace mvs {

texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat>
    ref_image_texture;
texture<uint8_t, hipTextureType2DLayered, hipReadModeNormalizedFloat>
    src_images_texture;
texture<float, hipTextureType2DLayered, hipReadModeElementType>
    src_depth_maps_texture;
texture<float, hipTextureType2D, hipReadModeElementType> poses_texture;

// Calibration of reference image (first two rows)
__constant__ float ref_K[6];
// Calibration of reference image ((first two rows))
__constant__ float ref_inv_K[6];

// Extrinsics of reference image in the scene coordinate frame
__constant__ float ref_R[9];
__constant__ float ref_T[3];
// projection center of the reference image in the scene coordinate frame
__constant__ float ref_C[3];

// 4 by 4 projection matrix of reference image and its inverse
// projection matrices are used to compute inter-image homography
__constant__ float ref_P[16];
__constant__ float ref_inv_P[16];

// minimum spatial resolution of these images
__constant__ float max_dist_per_pixel[1];


// homography
__device__ inline void HomographyWarp(const float mat[9],
                                      const float vec[2],
                                      float result[2]) {
  const float inv_third = 1.0f / (mat[6] * vec[0] + mat[7] * vec[1] + mat[8]);
  result[0] = inv_third * (mat[0] * vec[0] + mat[1] * vec[1] + mat[2]);
  result[1] = inv_third * (mat[3] * vec[0] + mat[4] * vec[1] + mat[5]);
}

// projection
__device__ inline void Projection(const float mat[16],
                                  const float vec[3],
                                  float result[3]) {
  const float inv_z = 1.0f / (mat[8] * vec[0] + mat[9] * vec[1] + mat[10] * vec[2] + mat[11]);
  result[0] = inv_z * (mat[0] * vec[0] + mat[1] * vec[1] + mat[2] * vec[2] + mat[3]);
  result[1] = inv_z * (mat[4] * vec[0] + mat[5] * vec[1] + mat[6] * vec[2] + mat[7]);
  // depth is now the fourth component
  result[2] = inv_z * (mat[12] * vec[0] + mat[13] * vec[1] + mat[14] * vec[2] + mat[15]);
}

// inverse projection
// depth is now defined as the fourth component
__device__ inline void InverseProjection(const float mat[16],
                                         const float vec[3],
                                         float result[3]) {
  const float depth = vec[2];
  const float inv_fourth = 1.0f / (mat[12] * vec[0] + mat[13] * vec[1] + mat[14] + mat[15] * depth);
  result[0] = inv_fourth * (mat[0] * vec[0] + mat[1] * vec[1] + mat[2] + mat[3] * depth);
  result[1] = inv_fourth * (mat[4] * vec[0] + mat[5] * vec[1] + mat[6] + mat[7] * depth);
  result[2] = inv_fourth * (mat[8] * vec[0] + mat[9] * vec[1] + mat[10] + mat[11] * depth);
}

// note that the returned point is in scene coordinate frame
// surface normal is also in scene coordinate frame
__device__ inline void ComputePointAtDepth(const float row, const float col,
                                           const float depth, float point[3]) {
  const float vec[3] = {col, row, depth};
  InverseProjection(ref_inv_P, vec, point);
  // for debug
//  printf("computepointatdepth: pixel: %f, %f, %f, point: %f, %f, %f\n", col, row, depth, point[0], point[1], point[2]);
}

__device__ inline float DotProduct3(const float vec1[3], const float vec2[3]) {
  return vec1[0] * vec2[0] + vec1[1] * vec2[1] + vec1[2] * vec2[2];
}

__device__ inline void CrossProduct3(const float vec1[3], const float vec2[3], float result[3]) {
  result[0] = vec1[1] * vec2[2] - vec1[2] * vec2[1];
  result[1] = vec1[2] * vec2[0] - vec1[0] * vec2[2];
  result[2] = vec1[0] * vec2[1] - vec1[1] * vec2[0];
}

// eucliden distance
__device__ inline float EuclidDist(const float vec1[3], const float vec2[3]) {
  const float diff[3] = {vec1[0] - vec2[0], vec1[1] - vec2[1], vec1[2] - vec2[2]};
  return sqrt(DotProduct3(diff, diff));
}

__device__ inline void Mat33DotVec3(const float mat[9], const float vec[3],
                                    float result[3]) {
  result[0] = mat[0] * vec[0] + mat[1] * vec[1] + mat[2] * vec[2];
  result[1] = mat[3] * vec[0] + mat[4] * vec[1] + mat[5] * vec[2];
  result[2] = mat[6] * vec[0] + mat[7] * vec[1] + mat[8] * vec[2];
}

//__device__ inline void Mat44DotVec4(const float mat[16], const float vec[4],
//                                    float result[4]) {
//  result[0] = mat[0] * vec[0] + mat[1] * vec[1] + mat[2] * vec[2] + mat[3] * vec[3];
//  result[1] = mat[4] * vec[0] + mat[5] * vec[1] + mat[6] * vec[2] + mat[7] * vec[3];
//  result[2] = mat[8] * vec[0] + mat[9] * vec[1] + mat[10] * vec[2] + mat[11] * vec[3];
//  result[3] = mat[12] * vec[0] + mat[13] * vec[1] + mat[14] * vec[2] + mat[15] * vec[3];
//}

__device__ inline void Vec4DotMat44(const float vec[4], const float mat[16],
                                    float result[4]) {
  result[0] = vec[0] * mat[0] + vec[1] * mat[4] + vec[2] * mat[8] + vec[3] * mat[12];
  result[1] = vec[0] * mat[1] + vec[1] * mat[5] + vec[2] * mat[9] + vec[3] * mat[13];
  result[2] = vec[0] * mat[2] + vec[1] * mat[6] + vec[2] * mat[10] + vec[3] * mat[14];
  result[3] = vec[0] * mat[3] + vec[1] * mat[7] + vec[2] * mat[11] + vec[3] * mat[15];
}

__device__ inline void Mat44DotMat44(const float mat1[16], const float mat2[16],
                                     float result[16]) {
  // first row
  result[0] = mat1[0] * mat2[0] + mat1[1] * mat2[4] + mat1[2] * mat2[8] + mat1[3] * mat2[12];
  result[1] = mat1[0] * mat2[1] + mat1[1] * mat2[5] + mat1[2] * mat2[9] + mat1[3] * mat2[13];
  result[2] = mat1[0] * mat2[2] + mat1[1] * mat2[6] + mat1[2] * mat2[10] + mat1[3] * mat2[14];
  result[3] = mat1[0] * mat2[3] + mat1[1] * mat2[7] + mat1[2] * mat2[11] + mat1[3] * mat2[15];

  // second row
  result[4] = mat1[4] * mat2[0] + mat1[5] * mat2[4] + mat1[6] * mat2[8] + mat1[7] * mat2[12];
  result[5] = mat1[4] * mat2[1] + mat1[5] * mat2[5] + mat1[6] * mat2[9] + mat1[7] * mat2[13];
  result[6] = mat1[4] * mat2[2] + mat1[5] * mat2[6] + mat1[6] * mat2[10] + mat1[7] * mat2[14];
  result[7] = mat1[4] * mat2[3] + mat1[5] * mat2[7] + mat1[6] * mat2[11] + mat1[7] * mat2[15];

  // third row
  result[8] = mat1[8] * mat2[0] + mat1[9] * mat2[4] + mat1[10] * mat2[8] + mat1[11] * mat2[12];
  result[9] = mat1[8] * mat2[1] + mat1[9] * mat2[5] + mat1[10] * mat2[9] + mat1[11] * mat2[13];
  result[10] = mat1[8] * mat2[2] + mat1[9] * mat2[6] + mat1[10] * mat2[10] + mat1[11] * mat2[14];
  result[11] = mat1[8] * mat2[3] + mat1[9] * mat2[7] + mat1[10] * mat2[11] + mat1[11] * mat2[15];

  // fourth row
  result[12] = mat1[12] * mat2[0] + mat1[13] * mat2[4] + mat1[14] * mat2[8] + mat1[15] * mat2[12];
  result[13] = mat1[12] * mat2[1] + mat1[13] * mat2[5] + mat1[14] * mat2[9] + mat1[15] * mat2[13];
  result[14] = mat1[12] * mat2[2] + mat1[13] * mat2[6] + mat1[14] * mat2[10] + mat1[15] * mat2[14];
  result[15] = mat1[12] * mat2[3] + mat1[13] * mat2[7] + mat1[14] * mat2[11] + mat1[15] * mat2[15];
}

__device__ inline float GenerateRandomDepth(const float depth_min,
                                            const float depth_max,
                                            hiprandState* rand_state) {
  return hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
}

// this might be improved by using non-uniform sampling
__device__ inline void GenerateRandomNormal(const int row, const int col,
                                            hiprandState* rand_state,
                                            float normal[3]) {
  // Unbiased sampling of normal, according to George Marsaglia, "Choosing a
  // Point from the Surface of a Sphere", 1972.
  float v1 = 0.0f;
  float v2 = 0.0f;
  float s = 2.0f;
  while (s >= 1.0f) {
    v1 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    v2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    s = v1 * v1 + v2 * v2;
  }

  const float s_norm = sqrt(1.0f - s);
  normal[0] = 2.0f * v1 * s_norm;
  normal[1] = 2.0f * v2 * s_norm;
  normal[2] = 1.0f - 2.0f * s;


  // make sure normal is pointing towards the camera
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1] * row + ref_inv_K[2],
                             ref_inv_K[3] * col + ref_inv_K[4] * row + ref_inv_K[5],
                             1.0f};
  // rotate view_ray to the scene coordinate frame
  // need a transpose of R
  float view_ray_scene[3];
  const float ref_R_transpose[9] = {ref_R[0], ref_R[3], ref_R[6],
                                    ref_R[1], ref_R[4], ref_R[7],
                                    ref_R[2], ref_R[5], ref_R[8]};
  Mat33DotVec3(ref_R_transpose, view_ray, view_ray_scene);


  if (DotProduct3(normal, view_ray_scene) >= 0.0f) {
    normal[0] = -normal[0];
    normal[1] = -normal[1];
    normal[2] = -normal[2];
  }
}

// make the perturbation more robust to big mean depth
__device__ inline float PerturbDepth(const float perturbation,
                                     const float global_depth_min,
                                     const float global_depth_max,
                                     const float depth,
                                     hiprandState* rand_state) {
  float depth_min = depth - perturbation * (global_depth_max - global_depth_min);
  float depth_max = depth + perturbation * (global_depth_max - global_depth_min);

  float depth_new = GenerateRandomDepth(depth_min, depth_max, rand_state);
  // clamp
  if (depth_new < global_depth_min) {
    depth_new = global_depth_min;
  }
  if (depth_new > global_depth_max) {
    depth_new = global_depth_max;
  }

  return depth_new;
}

// sampling from a cone that centers around the current normal vector
__device__ inline void PerturbNormal(const int row, const int col,
                                     const float max_perturbation_angle,
                                     const float normal[3],
                                     hiprandState* rand_state,
                                     float perturbed_normal[3],
                                     const int num_trials = 0) {

  // uniformly sample from a cone that centers around the normal vector

  // we first define a local coordinate frame whose z axis aligns with the normal direction
  // assume the z component of normal is always positive
  float local_x[3] = {0.0f, normal[2], -normal[1]};
  // normalize local_x
  const float inv_local_x_norm = rsqrt(DotProduct3(local_x, local_x));
  local_x[0] *= inv_local_x_norm;
  local_x[1] *= inv_local_x_norm;
  local_x[2] *= inv_local_x_norm;
  // compute local y direction as z\cross product x
  float local_y[3];
  CrossProduct3(normal, local_x, local_y);

  // generate a unit vector on the local x-y plane
  const float theta = hiprand_uniform(rand_state) * 2 * M_PI;
  const float cos_theta = cos(theta);
  const float sin_theta = sin(theta);
  // in the local coordinate frame, the vector has coordinate (cos theta, sin theta, 0)
  // we need to convert it back to the original coordinate frame
  // note that the rotation from the local coordinate frame to the original one is (local_x, local_y, local_z)
  const float vec[3] = { cos_theta * local_x[0] + sin_theta * local_y[0],
                         cos_theta * local_x[1] + sin_theta * local_y[1],
                         cos_theta * local_x[2] + sin_theta * local_y[2] };

  // compute cross product between vec and normal to get the rotation axis
  float rot_axis[3];
  CrossProduct3(vec, normal, rot_axis);

  // sample a perturbation angle around the rotation axis
  const float alpha = (hiprand_uniform(rand_state) - 0.5) * 2 * max_perturbation_angle;

  // the rotation matrix in the coordinate frame (vec, normal, rot_axis)
  //      is (cos alpha, -sin alpha, 0; sin alpha, cos alpha, 0; 0, 0, 1)
  // we need to represent this rotation in the original coordinate frame
  // the rotation from (vec, normal, rot_axis) to the original one is (vec, normal, rot_axis)
  // essentially by multiply (cos alpha, -sin alpha, 0; sin alpha, cos alpha, 0; 0, 0, 1) and
  //      (vec, normal, rot_axis)^T
  float R[9];
  const float cos_alpha = cos(alpha);
  const float sin_alpha = sin(alpha);
  R[0] = cos_alpha * vec[0] - sin_alpha * normal[0];
  R[1] = cos_alpha * vec[1] - sin_alpha * normal[1];
  R[2] = cos_alpha * vec[2] - sin_alpha * normal[2];

  R[3] = sin_alpha * vec[0] + cos_alpha * normal[0];
  R[4] = sin_alpha * vec[1] + cos_alpha * normal[1];
  R[5] = sin_alpha * vec[2] + cos_alpha * normal[2];

  R[6] = rot_axis[0];
  R[7] = rot_axis[1];
  R[8] = rot_axis[2];

  // Perturb the normal vector.
  Mat33DotVec3(R, normal, perturbed_normal);

  // Make sure the perturbed normal is still looking in the same direction as
  // the viewing direction, otherwise try again but with smaller perturbation.
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1] * row + ref_inv_K[2],
                             ref_inv_K[3] * col + ref_inv_K[4] * row + ref_inv_K[5],
                             1.0f};
  // rotate view_ray to the reference coordinate frame
  float view_ray_scene[3];
  const float ref_R_transpose[9] = {ref_R[0], ref_R[3], ref_R[6],
                                    ref_R[1], ref_R[4], ref_R[7],
                                    ref_R[2], ref_R[5], ref_R[8]};
  Mat33DotVec3(ref_R_transpose, view_ray, view_ray_scene);
  if (DotProduct3(perturbed_normal, view_ray_scene) >= 0.0f) {
    const int kMaxNumTrials = 3;
    if (num_trials < kMaxNumTrials) {
      PerturbNormal(row, col, 0.5f * max_perturbation_angle, normal, rand_state,
                    perturbed_normal, num_trials + 1);
      return;
    } else {
      perturbed_normal[0] = normal[0];
      perturbed_normal[1] = normal[1];
      perturbed_normal[2] = normal[2];
      // @ Sebastian
      // return;
    }
  }

  // Make sure normal has unit norm.
  const float inv_norm = rsqrt(DotProduct3(perturbed_normal, perturbed_normal));
  perturbed_normal[0] *= inv_norm;
  perturbed_normal[1] *= inv_norm;
  perturbed_normal[2] *= inv_norm;
}

// Transfer depth on plane from viewing ray at row1 to row2. The returned
// depth is the intersection of the viewing ray through row2 with the plane
// at row1 defined by the given depth and normal.
__device__ inline float PropagateDepth(const float depth1,
                                       const float normal1[3], const float col, const float row1,
                                       const float row2) {
  // first point
  float point1[3];
  ComputePointAtDepth(row1, col, depth1, point1);

  // collect co-efficients for the depth of pixel (col, row2)
  const float coeff = normal1[0] * (point1[0] * ref_inv_P[15] - ref_inv_P[3]) + \
                      normal1[1] * (point1[1] * ref_inv_P[15] - ref_inv_P[7]) + \
                      normal1[2] * (point1[2] * ref_inv_P[15] - ref_inv_P[11]);
  // collect rhs
  const float rhs =-( normal1[0] * (point1[0] * (ref_inv_P[12] * col + ref_inv_P[13] * row2 + ref_inv_P[14]) - ref_inv_P[0] * col - ref_inv_P[1] * row2 - ref_inv_P[2]) + \
                    normal1[1] * (point1[1] * (ref_inv_P[12] * col + ref_inv_P[13] * row2 + ref_inv_P[14]) - ref_inv_P[4] * col - ref_inv_P[5] * row2 - ref_inv_P[6]) + \
                    normal1[2] * (point1[2] * (ref_inv_P[12] * col + ref_inv_P[13] * row2 + ref_inv_P[14]) - ref_inv_P[8] * col - ref_inv_P[9] * row2 - ref_inv_P[10]) );
  // depth is now the fourth component
  float depth2 = rhs / coeff;

  // debug
//  printf("depth 1: %.6e, depth 2: %.6e\n", depth1, depth2);

  // make sure depth2 is not nan
  if (depth2 != depth2) {
    depth2 = depth1;
  }

  // double check the correctness
  float point2[3];
  ComputePointAtDepth(row2, col, depth2, point2);
  // if we deviate too much from point1, then there's some problem
  if (EuclidDist(point1, point2) > abs(row2 - row1) * max_dist_per_pixel[0]) {
    depth2 = depth1;
  }
  return depth2;
}

// First, compute triangulation angle between reference and source image for 3D
// point. Second, compute incident angle between viewing direction of source
// image and normal direction of 3D point. Both angles are cosine distances.
// Note that both point and normal coordinates are in scene coordinate frame
__device__ inline void ComputeViewingAngles(const float point[3],
                                            const float normal[3],
                                            const int image_idx,
                                            float* cos_triangulation_angle,
                                            float* cos_incident_angle) {
  *cos_triangulation_angle = 0.0f;
  *cos_incident_angle = 0.0f;

  // Projection center of source image.
  float src_C[3];
  for (int i = 0; i < 3; ++i) {
    src_C[i] = tex2D(poses_texture, i + 32, image_idx);
  }

  // Ray from point to reference camera
  const float RX[3] = {ref_C[0] - point[0], ref_C[1] - point[1], ref_C[2] - point[2]};
  // Ray from point to source camera
  const float SX[3] = {src_C[0] - point[0], src_C[1] - point[1], src_C[2] - point[2]};

  // Length of ray from reference image to point.
  const float RX_inv_norm = rsqrt(DotProduct3(RX, RX));

  // Length of ray from source image to point.
  const float SX_inv_norm = rsqrt(DotProduct3(SX, SX));

  *cos_incident_angle = DotProduct3(SX, normal) * SX_inv_norm;
  *cos_triangulation_angle = DotProduct3(RX, SX) * RX_inv_norm * SX_inv_norm;
}

// a more numerically stable way to compose homography
__device__ inline void ComposeHomography(const int image_idx, const int row,
                                         const int col, const float depth,
                                         const float normal[3], float H[9]) {
  // Extract projection matrices for source image.
  float P[16];
  for (int i = 0; i < 16; ++i) {
    P[i] = tex2D(poses_texture, i, image_idx);
  }

  // compute the plane n^Tx+c=0
  float point[3];
  ComputePointAtDepth(row, col, depth, point);
  const float c = -DotProduct3(point, normal);

  // compute the 1 by 4 vector [n; c]^T ref_inv_P
  float vec_tmp[4];
  const float plane[4] = {normal[0], normal[1], normal[2], c};
  Vec4DotMat44(plane, ref_inv_P, vec_tmp);

  // compute matrix P ref_inv_P
  float mat_tmp[16];
  Mat44DotMat44(P, ref_inv_P, mat_tmp);

  // the first three components of the fourth column of mat_tmp
  const float vec_a[3] = {-vec_tmp[0]/vec_tmp[3], -vec_tmp[1]/vec_tmp[3], -vec_tmp[2]/vec_tmp[3]};
  const float vec_b[3] = {mat_tmp[3], mat_tmp[7], mat_tmp[11]};
  const float mat_A[9] = {
      mat_tmp[0], mat_tmp[1], mat_tmp[2],
      mat_tmp[4], mat_tmp[5], mat_tmp[6],
      mat_tmp[8], mat_tmp[9], mat_tmp[10]
  };

  H[0] = mat_A[0] + vec_b[0] * vec_a[0];
  H[1] = mat_A[1] + vec_b[0] * vec_a[1];
  H[2] = mat_A[2] + vec_b[0] * vec_a[2];
  H[3] = mat_A[3] + vec_b[1] * vec_a[0];
  H[4] = mat_A[4] + vec_b[1] * vec_a[1];
  H[5] = mat_A[5] + vec_b[1] * vec_a[2];

  H[6] = mat_A[6] + vec_b[2] * vec_a[0];
  H[7] = mat_A[7] + vec_b[2] * vec_a[1];
  H[8] = mat_A[8] + vec_b[2] * vec_a[2];
}

// Each thread in the current warp / thread block reads in 3 columns of the
// reference image. The shared memory holds 3 * THREADS_PER_BLOCK columns and
// kWindowSize rows of the reference image. Each thread copies every
// THREADS_PER_BLOCK-th column from global to shared memory offset by its ID.
// For example, if THREADS_PER_BLOCK = 32, then thread 0 reads columns 0, 32, 64
// and thread 1 columns 1, 33, 65. When computing the photoconsistency, which is
// shared among each thread block, each thread can then read the reference image
// colors from shared memory. Note that this limits the window radius to a
// maximum of THREADS_PER_BLOCK.
template <int kWindowSize>
struct LocalRefImage {
  const static int kWindowRadius = kWindowSize / 2;
  const static int kThreadBlockRadius = 1;
  const static int kThreadBlockSize = 2 * kThreadBlockRadius + 1;
  const static int kNumRows = kWindowSize;
  const static int kNumColumns = kThreadBlockSize * THREADS_PER_BLOCK;
  const static int kDataSize = kNumRows * kNumColumns;

  float* data = nullptr;

  __device__ inline void Read(const int row) {
    // For the first row, read the entire block into shared memory. For all
    // consecutive rows, it is only necessary to shift the rows in shared memory
    // up by one element and then read in a new row at the bottom of the shared
    // memory. Note that this assumes that the calling loop starts with the
    // first row and then consecutively reads in the next row.

    const int thread_id = threadIdx.x;
    const int thread_block_first_id = blockDim.x * blockIdx.x;

    const int local_col_start = thread_id;
    const int global_col_start = thread_block_first_id -
                                 kThreadBlockRadius * THREADS_PER_BLOCK +
                                 thread_id;

    if (row == 0) {
      int global_row = row - kWindowRadius;
      for (int local_row = 0; local_row < kNumRows; ++local_row, ++global_row) {
        int local_col = local_col_start;
        int global_col = global_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[local_row * kNumColumns + local_col] =
              tex2D(ref_image_texture, global_col, global_row);
          local_col += THREADS_PER_BLOCK;
          global_col += THREADS_PER_BLOCK;
        }
      }
    } else {
      // Move rows in shared memory up by one row.
      for (int local_row = 1; local_row < kNumRows; ++local_row) {
        int local_col = local_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[(local_row - 1) * kNumColumns + local_col] =
              data[local_row * kNumColumns + local_col];
          local_col += THREADS_PER_BLOCK;
        }
      }

      // Read next row into the last row of shared memory.
      const int local_row = kNumRows - 1;
      const int global_row = row + kWindowRadius;
      int local_col = local_col_start;
      int global_col = global_col_start;
#pragma unroll
      for (int block = 0; block < kThreadBlockSize; ++block) {
        data[local_row * kNumColumns + local_col] =
            tex2D(ref_image_texture, global_col, global_row);
        local_col += THREADS_PER_BLOCK;
        global_col += THREADS_PER_BLOCK;
      }
    }
  }
};

// The return values is 1 - NCC, so the range is [0, 2], the smaller the
// value, the better the color consistency.
template <int kWindowSize, int kWindowStep>
struct PhotoConsistencyCostComputer {
  const static int kWindowRadius = kWindowSize / 2;

  __device__ PhotoConsistencyCostComputer(const float sigma_spatial,
                                          const float sigma_color)
      : bilateral_weight_computer_(sigma_spatial, sigma_color) {}

  // Maximum photo consistency cost as 1 - min(NCC).
  const float kMaxCost = 2.0f;

  // Thread warp local reference image data around current patch.
  typedef LocalRefImage<kWindowSize> LocalRefImageType;
  LocalRefImageType local_ref_image;

  // Precomputed sum of raw and squared image intensities.
  float local_ref_sum = 0.0f;
  float local_ref_squared_sum = 0.0f;

  // Index of source image.
  int src_image_idx = -1;

  // Center position of patch in reference image.
  int row = -1;
  int col = -1;

  // Depth and normal for which to warp patch.
  float depth = -1e20f;
  const float* normal = nullptr;

  __device__ inline void Read(const int row) {
    local_ref_image.Read(row);
    __syncthreads();
  }

  __device__ inline float Compute() const {
    float tform[9];
    ComposeHomography(src_image_idx, row, col, depth, normal, tform);

    float tform_step[8];
    for (int i = 0; i < 8; ++i) {
      tform_step[i] = kWindowStep * tform[i];
    }

    const int thread_id = threadIdx.x;
    const int row_start = row - kWindowRadius;
    const int col_start = col - kWindowRadius;

    float col_src = tform[0] * col_start + tform[1] * row_start + tform[2];
    float row_src = tform[3] * col_start + tform[4] * row_start + tform[5];
    float z = tform[6] * col_start + tform[7] * row_start + tform[8];
    float base_col_src = col_src;
    float base_row_src = row_src;
    float base_z = z;

    int ref_image_idx = THREADS_PER_BLOCK - kWindowRadius + thread_id;
    int ref_image_base_idx = ref_image_idx;

    const float ref_center_color =
        local_ref_image
            .data[ref_image_idx + kWindowRadius * 3 * THREADS_PER_BLOCK +
                  kWindowRadius];
    const float ref_color_sum = local_ref_sum;
    const float ref_color_squared_sum = local_ref_squared_sum;
    float src_color_sum = 0.0f;
    float src_color_squared_sum = 0.0f;
    float src_ref_color_sum = 0.0f;
    float bilateral_weight_sum = 0.0f;

    for (int row = -kWindowRadius; row <= kWindowRadius; row += kWindowStep) {
      for (int col = -kWindowRadius; col <= kWindowRadius; col += kWindowStep) {
        const float inv_z = 1.0f / z;
        const float norm_col_src = inv_z * col_src + 0.5f;  // half pixel is due to GPU's texture memory
        const float norm_row_src = inv_z * row_src + 0.5f;
        const float ref_color = local_ref_image.data[ref_image_idx];
        const float src_color = tex2DLayered(src_images_texture, norm_col_src,
                                             norm_row_src, src_image_idx);

        const float bilateral_weight = bilateral_weight_computer_.Compute(
            row, col, ref_center_color, ref_color);

        const float bilateral_weight_src = bilateral_weight * src_color;

        src_color_sum += bilateral_weight_src;
        src_color_squared_sum += bilateral_weight_src * src_color;
        src_ref_color_sum += bilateral_weight_src * ref_color;
        bilateral_weight_sum += bilateral_weight;

        ref_image_idx += kWindowStep;

        // Accumulate warped source coordinates per row to reduce numerical
        // errors. Note that this is necessary since coordinates usually are in
        // the order of 1000s as opposed to the color values which are
        // normalized to the range [0, 1].
        col_src += tform_step[0];
        row_src += tform_step[3];
        z += tform_step[6];
      }

      ref_image_base_idx += kWindowStep * 3 * THREADS_PER_BLOCK;
      ref_image_idx = ref_image_base_idx;

      base_col_src += tform_step[1];
      base_row_src += tform_step[4];
      base_z += tform_step[7];

      col_src = base_col_src;
      row_src = base_row_src;
      z = base_z;
    }

    const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
    src_color_sum *= inv_bilateral_weight_sum;
    src_color_squared_sum *= inv_bilateral_weight_sum;
    src_ref_color_sum *= inv_bilateral_weight_sum;

    const float ref_color_var =
        ref_color_squared_sum - ref_color_sum * ref_color_sum;
    const float src_color_var =
        src_color_squared_sum - src_color_sum * src_color_sum;

    // Based on Jensen's Inequality for convex functions, the variance
    // should always be larger than 0. Do not make this threshold smaller.
    constexpr float kMinVar = 1e-5f;
    if (ref_color_var < kMinVar || src_color_var < kMinVar) {
      return kMaxCost;
    } else {
      const float src_ref_color_covar =
          src_ref_color_sum - ref_color_sum * src_color_sum;
      const float src_ref_color_var = sqrt(ref_color_var * src_color_var);
      return max(0.0f,
                 min(kMaxCost, 1.0f - src_ref_color_covar / src_ref_color_var));
    }
  }

 private:
  const BilateralWeightComputer bilateral_weight_computer_;
};

// important
__device__ inline float ComputeGeomConsistencyCost(const float row,
                                                   const float col,
                                                   const float depth,
                                                   const int image_idx,
                                                   const float max_cost) {
  // Extract projection matrices for source image.
  float P[16];
  for (int i = 0; i < 16; ++i) {
    P[i] = tex2D(poses_texture, i, image_idx);
  }
  float inv_P[16];
  for (int i = 0; i < 16; ++i) {
    inv_P[i] = tex2D(poses_texture, i + 16, image_idx);
  }

  // Project point in reference image to world.
  float forward_point[3];
  ComputePointAtDepth(row, col, depth, forward_point);

  // Project world point to source image.
  float src_pixel[2];
  Projection(P, forward_point, src_pixel);

  // Extract depth in source image.
  // why would we need a half pixel here
  const float src_depth = tex2DLayered(src_depth_maps_texture, src_pixel[0] + 0.5f,
                                       src_pixel[1] + 0.5f, image_idx);

  // Projection outside of source image.
  if (src_depth <= -1e19f) {
    return max_cost;
  }

  // Project point in source image to world.
  float backward_point[3];
  const float src_pixel_depth[3] = {src_pixel[0], src_pixel[1], src_depth};
  InverseProjection(inv_P, src_pixel_depth, backward_point);

  // Project world point back to reference image.
  float ref_pixel[2];
  Projection(ref_P, backward_point, ref_pixel);

  // Return truncated reprojection error between original observation and
  // the forward-backward projected observation.
  const float diff_col = col - ref_pixel[0];
  const float diff_row = row - ref_pixel[1];
  return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

// Find index of minimum in given values.
template <int kNumCosts>
__device__ inline int FindMinCost(const float costs[kNumCosts]) {
  float min_cost = costs[0];
  int min_cost_idx = 0;
  for (int idx = 1; idx < kNumCosts; ++idx) {
    if (costs[idx] <= min_cost) {
      min_cost = costs[idx];
      min_cost_idx = idx;
    }
  }
  return min_cost_idx;
}

__device__ inline void TransformPDFToCDF(float* probs, const int num_probs) {
  float prob_sum = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    prob_sum += probs[i];
  }
  const float inv_prob_sum = 1.0f / prob_sum;

  float cum_prob = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    const float prob = probs[i] * inv_prob_sum;
    cum_prob += prob;
    probs[i] = cum_prob;
  }
}

class LikelihoodComputer {
 public:
  __device__ LikelihoodComputer(const float ncc_sigma,
                                const float min_triangulation_angle,
                                const float incident_angle_sigma)
      : cos_min_triangulation_angle_(cos(min_triangulation_angle)),
        inv_incident_angle_sigma_square_(
            -0.5f / (incident_angle_sigma * incident_angle_sigma)),
        inv_ncc_sigma_square_(-0.5f / (ncc_sigma * ncc_sigma)),
        ncc_norm_factor_(ComputeNCCCostNormFactor(ncc_sigma)) {}

  // Compute forward message from current cost and forward message of
  // previous / neighboring pixel.
  __device__ float ComputeForwardMessage(const float cost,
                                         const float prev) const {
    return ComputeMessage<true>(cost, prev);
  }

  // Compute backward message from current cost and backward message of
  // previous / neighboring pixel.
  __device__ float ComputeBackwardMessage(const float cost,
                                          const float prev) const {
    return ComputeMessage<false>(cost, prev);
  }

  // Compute the selection probability from the forward and backward message.
  __device__ inline float ComputeSelProb(const float alpha, const float beta,
                                         const float prev,
                                         const float prev_weight) const {
    const float zn0 = (1.0f - alpha) * (1.0f - beta);
    const float zn1 = alpha * beta;
    const float curr = zn1 / (zn0 + zn1);
    return prev_weight * prev + (1.0f - prev_weight) * curr;
  }

  // Compute NCC probability. Note that cost = 1 - NCC.
  __device__ inline float ComputeNCCProb(const float cost) const {
    return exp(cost * cost * inv_ncc_sigma_square_) * ncc_norm_factor_;
  }

  // Compute the triangulation angle probability.
  __device__ inline float ComputeTriProb(
      const float cos_triangulation_angle) const {
    const float abs_cos_triangulation_angle = abs(cos_triangulation_angle);
    if (abs_cos_triangulation_angle > cos_min_triangulation_angle_) {
      const float scaled = 1.0f - (1.0f - abs_cos_triangulation_angle) /
                                      (1.0f - cos_min_triangulation_angle_);
      const float likelihood = 1.0f - scaled * scaled;
      return min(1.0f, max(0.0f, likelihood));
    } else {
      return 1.0f;
    }
  }

  // Compute the incident angle probability.
  __device__ inline float ComputeIncProb(const float cos_incident_angle) const {
    const float x = 1.0f - max(0.0f, cos_incident_angle);
    return exp(x * x * inv_incident_angle_sigma_square_);
  }

  // Compute the warping/resolution prior probability.
  template <int kWindowSize>
  __device__ inline float ComputeResolutionProb(const float H[9],
                                                const float row,
                                                const float col) const {
    const int kWindowRadius = kWindowSize / 2;

    // Warp corners of patch in reference image to source image.
    float src1[2];
    const float ref1[2] = {col - kWindowRadius, row - kWindowRadius};
    HomographyWarp(H, ref1, src1);
    float src2[2];
    const float ref2[2] = {col - kWindowRadius, row + kWindowRadius};
    HomographyWarp(H, ref2, src2);
    float src3[2];
    const float ref3[2] = {col + kWindowRadius, row + kWindowRadius};
    HomographyWarp(H, ref3, src3);
    float src4[2];
    const float ref4[2] = {col + kWindowRadius, row - kWindowRadius};
    HomographyWarp(H, ref4, src4);

    // Compute area of patches in reference and source image.
    const float ref_area = kWindowSize * kWindowSize;
    const float src_area =
        abs(0.5f * (src1[0] * src2[1] - src2[0] * src1[1] - src1[0] * src4[1] +
                    src2[0] * src3[1] - src3[0] * src2[1] + src4[0] * src1[1] +
                    src3[0] * src4[1] - src4[0] * src3[1]));

    if (ref_area > src_area) {
      return src_area / ref_area;
    } else {
      return ref_area / src_area;
    }
  }

 private:
  // The normalization for the likelihood function, i.e. the normalization for
  // the prior on the matching cost.
  __device__ static inline float ComputeNCCCostNormFactor(
      const float ncc_sigma) {
    // A = sqrt(2pi)*sigma/2*erf(sqrt(2)/sigma)
    // erf(x) = 2/sqrt(pi) * integral from 0 to x of exp(-t^2) dt
    return 2.0f / (sqrt(2.0f * M_PI) * ncc_sigma *
                   erff(2.0f / (ncc_sigma * 1.414213562f)));
  }

  // Compute the forward or backward message.
  template <bool kForward>
  __device__ inline float ComputeMessage(const float cost,
                                         const float prev) const {
    constexpr float kUniformProb = 0.5f;
    constexpr float kNoChangeProb = 0.99999f;
    const float kChangeProb = 1.0f - kNoChangeProb;
    const float emission = ComputeNCCProb(cost);

    float zn0;  // Message for selection probability = 0.
    float zn1;  // Message for selection probability = 1.
    if (kForward) {
      zn0 = (prev * kChangeProb + (1.0f - prev) * kNoChangeProb) * kUniformProb;
      zn1 = (prev * kNoChangeProb + (1.0f - prev) * kChangeProb) * emission;
    } else {
      zn0 = prev * emission * kChangeProb +
            (1.0f - prev) * kUniformProb * kNoChangeProb;
      zn1 = prev * emission * kNoChangeProb +
            (1.0f - prev) * kUniformProb * kChangeProb;
    }

    return zn1 / (zn0 + zn1);
  }

  float cos_min_triangulation_angle_;
  float inv_incident_angle_sigma_square_;
  float inv_ncc_sigma_square_;
  float ncc_norm_factor_;
};

__global__ void InitNormalMap(GpuMat<float> normal_map,
                              GpuMat<hiprandState> rand_state_map) {
  const int row = blockDim.y * blockIdx.y + threadIdx.y;
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  if (col < normal_map.GetWidth() && row < normal_map.GetHeight()) {
    hiprandState rand_state = rand_state_map.Get(row, col);
    float normal[3];
    GenerateRandomNormal(row, col, &rand_state, normal);
    normal_map.SetSlice(row, col, normal);
    rand_state_map.Set(row, col, rand_state);
  }
}

template <int kWindowSize, int kWindowStep>
__global__ void ComputeInitialCost(GpuMat<float> cost_map,
                                   const GpuMat<float> depth_map,
                                   const GpuMat<float> normal_map,
                                   const GpuMat<float> ref_sum_image,
                                   const GpuMat<float> ref_squared_sum_image,
                                   const float sigma_spatial,
                                   const float sigma_color) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(sigma_spatial, sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  float normal[3] = {0};
  pcc_computer.normal = normal;

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col < cost_map.GetWidth()) {
      pcc_computer.depth = depth_map.Get(row, col);
      normal_map.GetSlice(row, col, normal);

      pcc_computer.row = row;
      pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
      pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        pcc_computer.src_image_idx = image_idx;
        cost_map.Set(row, col, image_idx, pcc_computer.Compute());
      }
    }
  }
}

struct SweepOptions {
  float perturbation = 1.0f;
  float depth_min = 0.0f;
  float depth_max = 1.0f;
  int num_samples = 15;
  float sigma_spatial = 3.0f;
  float sigma_color = 0.3f;
  float ncc_sigma = 0.6f;
  float min_triangulation_angle = 0.5f;
  float incident_angle_sigma = 0.9f;
  float prev_sel_prob_weight = 0.0f;
  float geom_consistency_regularizer = 0.1f;
  float geom_consistency_max_cost = 5.0f;
  float filter_min_ncc = 0.1f;
  float filter_min_triangulation_angle = 3.0f;
  int filter_min_num_consistent = 2;
  float filter_geom_consistency_max_cost = 1.0f;
};

template <int kWindowSize, int kWindowStep, bool kGeomConsistencyTerm = false,
          bool kFilterPhotoConsistency = false,
          bool kFilterGeomConsistency = false>
__global__ void SweepFromTopToBottom(
    GpuMat<float> global_workspace, GpuMat<hiprandState> rand_state_map,
    GpuMat<float> cost_map, GpuMat<float> depth_map, GpuMat<float> normal_map,
    GpuMat<uint8_t> consistency_mask, GpuMat<float> sel_prob_map,
    const GpuMat<float> prev_sel_prob_map, const GpuMat<float> ref_sum_image,
    const GpuMat<float> ref_squared_sum_image, const SweepOptions options) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  // Probability for boundary pixels.
  constexpr float kUniformProb = 0.5f;

  LikelihoodComputer likelihood_computer(options.ncc_sigma,
                                         options.min_triangulation_angle,
                                         options.incident_angle_sigma);

  float* forward_message =
      &global_workspace.GetPtr()[col * global_workspace.GetHeight()];
  float* sampling_probs =
      &global_workspace.GetPtr()[global_workspace.GetWidth() *
                                     global_workspace.GetHeight() +
                                 col * global_workspace.GetHeight()];

  //////////////////////////////////////////////////////////////////////////////
  // Compute backward message for all rows. Note that the backward messages are
  // temporarily stored in the sel_prob_map and replaced row by row as the
  // updated forward messages are computed further below.
  //////////////////////////////////////////////////////////////////////////////

  if (col < cost_map.GetWidth()) {
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Compute backward message.
      float beta = kUniformProb;
      for (int row = cost_map.GetHeight() - 1; row >= 0; --row) {
        const float cost = cost_map.Get(row, col, image_idx);
        beta = likelihood_computer.ComputeBackwardMessage(cost, beta);
        sel_prob_map.Set(row, col, image_idx, beta);
      }

      // Initialize forward message.
      forward_message[image_idx] = kUniformProb;
    }
  }

  //////////////////////////////////////////////////////////////////////////////
  // Estimate parameters for remaining rows and compute selection probabilities.
  //////////////////////////////////////////////////////////////////////////////

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(options.sigma_spatial,
                                                options.sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  struct ParamState {
    float depth = -1e20f;  // absurd value
    float normal[3] = {0};
  };

  // Parameters of previous pixel in column.
  ParamState prev_param_state;
  // Parameters of current pixel in column.
  ParamState curr_param_state;
  // Randomly sampled parameters.
  ParamState rand_param_state;
  // Cuda PRNG state for random sampling.
  hiprandState rand_state;

  if (col < cost_map.GetWidth()) {
    // Read random state for current column.
    rand_state = rand_state_map.Get(0, col);
    // Parameters for first row in column.
    prev_param_state.depth = depth_map.Get(0, col);
    normal_map.GetSlice(0, col, prev_param_state.normal);
  }

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col >= cost_map.GetWidth()) {
      continue;
    }

    pcc_computer.row = row;
    pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
    pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

    // Propagate the depth at which the current ray intersects with the plane
    // of the normal of the previous ray. This helps to better estimate
    // the depth of very oblique structures, i.e. pixels whose normal direction
    // is significantly different from their viewing direction.
    prev_param_state.depth = PropagateDepth(
        prev_param_state.depth, prev_param_state.normal, col, row - 1, row);

    // Read parameters for current pixel from previous sweep.
    curr_param_state.depth = depth_map.Get(row, col);
    normal_map.GetSlice(row, col, curr_param_state.normal);

    // Generate random parameters.
    rand_param_state.depth =
        PerturbDepth(options.perturbation, options.depth_min, options.depth_max, curr_param_state.depth, &rand_state);
    PerturbNormal(row, col, options.perturbation * M_PI,
                  curr_param_state.normal, &rand_state,
                  rand_param_state.normal);

    // Read in the backward message, compute selection probabilities and
    // modulate selection probabilities with priors.

    float point[3];
    ComputePointAtDepth(row, col, curr_param_state.depth, point);

    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      const float cost = cost_map.Get(row, col, image_idx);
      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float sel_prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);

      float cos_triangulation_angle;
      float cos_incident_angle;
      ComputeViewingAngles(point, curr_param_state.normal, image_idx,
                           &cos_triangulation_angle, &cos_incident_angle);
      const float tri_prob =
          likelihood_computer.ComputeTriProb(cos_triangulation_angle);
      const float inc_prob =
          likelihood_computer.ComputeIncProb(cos_incident_angle);

      float H[9];
      ComposeHomography(image_idx, row, col, curr_param_state.depth,
                        curr_param_state.normal, H);
      const float res_prob =
          likelihood_computer.ComputeResolutionProb<kWindowSize>(H, row, col);

      sampling_probs[image_idx] = sel_prob * tri_prob * inc_prob * res_prob;
    }

    TransformPDFToCDF(sampling_probs, cost_map.GetDepth());

    // Compute matching cost using Monte Carlo sampling of source images. Images
    // with higher selection probability are more likely to be sampled. Hence,
    // if only very few source images see the reference image pixel, the same
    // source image is likely to be sampled many times. Instead of taking
    // the best K probabilities, this sampling scheme has the advantage of
    // being adaptive to any distribution of selection probabilities.

    constexpr int kNumCosts = 5;
    float costs[kNumCosts] = {0};
    const float depths[kNumCosts] = {
        curr_param_state.depth, prev_param_state.depth, rand_param_state.depth,
        curr_param_state.depth, rand_param_state.depth};
    const float* normals[kNumCosts] = {
        curr_param_state.normal, prev_param_state.normal,
        rand_param_state.normal, rand_param_state.normal,
        curr_param_state.normal};

    for (int sample = 0; sample < options.num_samples; ++sample) {
      const float rand_prob = hiprand_uniform(&rand_state) - FLT_EPSILON;

      pcc_computer.src_image_idx = -1;
      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        const float prob = sampling_probs[image_idx];
        if (prob > rand_prob) {
          pcc_computer.src_image_idx = image_idx;
          break;
        }
      }

      if (pcc_computer.src_image_idx == -1) {
        continue;
      }

      costs[0] += cost_map.Get(row, col, pcc_computer.src_image_idx);
      if (kGeomConsistencyTerm) {
        costs[0] += options.geom_consistency_regularizer *
                    ComputeGeomConsistencyCost(
                        row, col, depths[0], pcc_computer.src_image_idx,
                        options.geom_consistency_max_cost);
      }

      for (int i = 1; i < kNumCosts; ++i) {
        pcc_computer.depth = depths[i];
        pcc_computer.normal = normals[i];
        costs[i] += pcc_computer.Compute();
        if (kGeomConsistencyTerm) {
          costs[i] += options.geom_consistency_regularizer *
                      ComputeGeomConsistencyCost(
                          row, col, depths[i], pcc_computer.src_image_idx,
                          options.geom_consistency_max_cost);
        }
      }
    }

    // Find the parameters of the minimum cost.
    const int min_cost_idx = FindMinCost<kNumCosts>(costs);
    const float best_depth = depths[min_cost_idx];
    const float* best_normal = normals[min_cost_idx];

    // Save best new parameters.
    depth_map.Set(row, col, best_depth);
    normal_map.SetSlice(row, col, best_normal);

    // Use the new cost to recompute the updated forward message and
    // the selection probability.
    pcc_computer.depth = best_depth;
    pcc_computer.normal = best_normal;
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Determine the cost for best depth.
      float cost;
      if (min_cost_idx == 0) {
        cost = cost_map.Get(row, col, image_idx);
      } else {
        pcc_computer.src_image_idx = image_idx;
        cost = pcc_computer.Compute();
        cost_map.Set(row, col, image_idx, cost);
      }

      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);
      forward_message[image_idx] = alpha;
      sel_prob_map.Set(row, col, image_idx, prob);
    }

    if (kFilterPhotoConsistency || kFilterGeomConsistency) {
      int num_consistent = 0;

      float best_point[3];
      ComputePointAtDepth(row, col, best_depth, best_point);

      const float min_ncc_prob =
          likelihood_computer.ComputeNCCProb(1.0f - options.filter_min_ncc);
      const float cos_min_triangulation_angle =
          cos(options.filter_min_triangulation_angle);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        float cos_triangulation_angle;
        float cos_incident_angle;
        ComputeViewingAngles(best_point, best_normal, image_idx,
                             &cos_triangulation_angle, &cos_incident_angle);
        // triangulation angle should not be too big or too small
        // which is why we need to take the absolute value
        if (abs(cos_triangulation_angle) > cos_min_triangulation_angle ||
            cos_incident_angle <= 0.0f) {
          continue;
        }

        if (!kFilterGeomConsistency) {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else if (!kFilterPhotoConsistency) {
          if (ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
              options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob &&
              ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
                  options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        }
      }

      if (num_consistent < options.filter_min_num_consistent) {
        const float kFilterValue = -1e20f;
        depth_map.Set(row, col, kFilterValue);
        normal_map.Set(row, col, 0, kFilterValue);
        normal_map.Set(row, col, 1, kFilterValue);
        normal_map.Set(row, col, 2, kFilterValue);
        for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
          consistency_mask.Set(row, col, image_idx, 0);
        }
      }
    }

    // Update previous depth for next row.
    prev_param_state.depth = best_depth;
    for (int i = 0; i < 3; ++i) {
      prev_param_state.normal[i] = best_normal[i];
    }
  }

  if (col < cost_map.GetWidth()) {
    rand_state_map.Set(0, col, rand_state);
  }
}

PatchMatchCuda::PatchMatchCuda(const PatchMatchOptions& options,
                               const PatchMatch::Problem& problem)
    : options_(options),
      problem_(problem),
      ref_width_(0),
      ref_height_(0),
      rotation_in_half_pi_(0) {
  SetBestCudaDevice(std::stoi(options_.gpu_index));
  InitRefImage();
  InitSourceImages();
  InitTransforms();
  InitWorkspaceMemory();
}

PatchMatchCuda::~PatchMatchCuda() {
  poses_device_.reset();
}

void PatchMatchCuda::Run() {
#define CASE_WINDOW_RADIUS(window_radius, window_step)              \
  case window_radius:                                               \
    RunWithWindowSizeAndStep<2 * window_radius + 1, window_step>(); \
    break;

#define CASE_WINDOW_STEP(window_step)                                 \
  case window_step:                                                   \
    switch (options_.window_radius) {                                 \
      CASE_WINDOW_RADIUS(1, window_step)                              \
      CASE_WINDOW_RADIUS(2, window_step)                              \
      CASE_WINDOW_RADIUS(3, window_step)                              \
      CASE_WINDOW_RADIUS(4, window_step)                              \
      CASE_WINDOW_RADIUS(5, window_step)                              \
      CASE_WINDOW_RADIUS(6, window_step)                              \
      CASE_WINDOW_RADIUS(7, window_step)                              \
      CASE_WINDOW_RADIUS(8, window_step)                              \
      CASE_WINDOW_RADIUS(9, window_step)                              \
      CASE_WINDOW_RADIUS(10, window_step)                             \
      CASE_WINDOW_RADIUS(11, window_step)                             \
      CASE_WINDOW_RADIUS(12, window_step)                             \
      CASE_WINDOW_RADIUS(13, window_step)                             \
      CASE_WINDOW_RADIUS(14, window_step)                             \
      CASE_WINDOW_RADIUS(15, window_step)                             \
      CASE_WINDOW_RADIUS(16, window_step)                             \
      CASE_WINDOW_RADIUS(17, window_step)                             \
      CASE_WINDOW_RADIUS(18, window_step)                             \
      CASE_WINDOW_RADIUS(19, window_step)                             \
      CASE_WINDOW_RADIUS(20, window_step)                             \
      default: {                                                      \
        std::cerr << "Error: Window size not supported" << std::endl; \
        break;                                                        \
      }                                                               \
    }                                                                 \
    break;

  switch (options_.window_step) {
    CASE_WINDOW_STEP(1)
    CASE_WINDOW_STEP(2)
    default: {
      std::cerr << "Error: Window step not supported" << std::endl;
      break;
    }
  }

#undef SWITCH_WINDOW_RADIUS
#undef CALL_RUN_FUNC
}

DepthMap PatchMatchCuda::GetDepthMap() const {
  return DepthMap(depth_map_->CopyToMat(), options_.depth_min,
                  options_.depth_max);
}

NormalMap PatchMatchCuda::GetNormalMap() const {
  return NormalMap(normal_map_->CopyToMat());
}

Mat<float> PatchMatchCuda::GetSelProbMap() const {
  return prev_sel_prob_map_->CopyToMat();
}

std::vector<int> PatchMatchCuda::GetConsistentImageIdxs() const {
  const Mat<uint8_t> mask = consistency_mask_->CopyToMat();
  std::vector<int> consistent_image_idxs;
  std::vector<int> pixel_consistent_image_idxs;
  pixel_consistent_image_idxs.reserve(mask.GetDepth());
  for (size_t r = 0; r < mask.GetHeight(); ++r) {
    for (size_t c = 0; c < mask.GetWidth(); ++c) {
      pixel_consistent_image_idxs.clear();
      for (size_t d = 0; d < mask.GetDepth(); ++d) {
        if (mask.Get(r, c, d)) {
          pixel_consistent_image_idxs.push_back(problem_.src_image_idxs[d]);
        }
      }
      if (pixel_consistent_image_idxs.size() > 0) {
        consistent_image_idxs.push_back(c);
        consistent_image_idxs.push_back(r);
        consistent_image_idxs.push_back(pixel_consistent_image_idxs.size());
        consistent_image_idxs.insert(consistent_image_idxs.end(),
                                     pixel_consistent_image_idxs.begin(),
                                     pixel_consistent_image_idxs.end());
      }
    }
  }
  return consistent_image_idxs;
}

template <int kWindowSize, int kWindowStep>
void PatchMatchCuda::RunWithWindowSizeAndStep() {
  // Wait for all initializations to finish.
  CUDA_SYNC_AND_CHECK();

  CudaTimer total_timer;
  CudaTimer init_timer;

  ComputeCudaConfig();
  ComputeInitialCost<kWindowSize, kWindowStep>
      <<<sweep_grid_size_, sweep_block_size_>>>(
          *cost_map_, *depth_map_, *normal_map_, *ref_image_->sum_image,
          *ref_image_->squared_sum_image, options_.sigma_spatial,
          options_.sigma_color);
  CUDA_SYNC_AND_CHECK();

  init_timer.Print("Initialization");

  const float total_num_steps = options_.num_iterations * 4;

  SweepOptions sweep_options;
  sweep_options.depth_min = options_.depth_min;
  sweep_options.depth_max = options_.depth_max;
  sweep_options.sigma_spatial = options_.sigma_spatial;
  sweep_options.sigma_color = options_.sigma_color;
  sweep_options.num_samples = options_.num_samples;
  sweep_options.ncc_sigma = options_.ncc_sigma;
  sweep_options.min_triangulation_angle =
      DEG2RAD(options_.min_triangulation_angle);
  sweep_options.incident_angle_sigma = options_.incident_angle_sigma;
  sweep_options.geom_consistency_regularizer =
      options_.geom_consistency_regularizer;
  sweep_options.geom_consistency_max_cost = options_.geom_consistency_max_cost;
  sweep_options.filter_min_ncc = options_.filter_min_ncc;
  sweep_options.filter_min_triangulation_angle =
      DEG2RAD(options_.filter_min_triangulation_angle);
  sweep_options.filter_min_num_consistent = options_.filter_min_num_consistent;
  sweep_options.filter_geom_consistency_max_cost =
      options_.filter_geom_consistency_max_cost;

  for (int iter = 0; iter < options_.num_iterations; ++iter) {
    CudaTimer iter_timer;

    for (int sweep = 0; sweep < 4; ++sweep) {
      CudaTimer sweep_timer;

      // Expenentially reduce amount of perturbation during the optimization.
      sweep_options.perturbation = 1.0f / std::pow(2.0f, iter + sweep / 4.0f);

      // Linearly increase the influence of previous selection probabilities.
      sweep_options.prev_sel_prob_weight =
          static_cast<float>(iter * 4 + sweep) / total_num_steps;

      const bool last_sweep = iter == options_.num_iterations - 1 && sweep == 3;

//      printf("\nsweep: %i", sweep);
//      int numBlock = 1;
//      int numThreadsPerBlock = 1;
//      PrintSetting<<<numBlock,numThreadsPerBlock>>>();

#define CALL_SWEEP_FUNC                                                  \
  SweepFromTopToBottom<kWindowSize, kWindowStep, kGeomConsistencyTerm,   \
                       kFilterPhotoConsistency, kFilterGeomConsistency>  \
      <<<sweep_grid_size_, sweep_block_size_>>>(                         \
          *global_workspace_, *rand_state_map_, *cost_map_, *depth_map_, \
          *normal_map_, *consistency_mask_, *sel_prob_map_,              \
          *prev_sel_prob_map_, *ref_image_->sum_image,                   \
          *ref_image_->squared_sum_image, sweep_options);

      if (last_sweep) {
        if (options_.filter) {
          consistency_mask_.reset(new GpuMat<uint8_t>(cost_map_->GetWidth(),
                                                      cost_map_->GetHeight(),
                                                      cost_map_->GetDepth()));
          consistency_mask_->FillWithScalar(0);
        }
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = true;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        } else {
          const bool kGeomConsistencyTerm = false;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        }
      } else {
        const bool kFilterPhotoConsistency = false;
        const bool kFilterGeomConsistency = false;
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          CALL_SWEEP_FUNC
        } else {
          const bool kGeomConsistencyTerm = false;
          CALL_SWEEP_FUNC
        }
      }

#undef CALL_SWEEP_FUNC

      CUDA_SYNC_AND_CHECK();

      Rotate();

      // Rotate selected image map.
      if (last_sweep && options_.filter) {
        std::unique_ptr<GpuMat<uint8_t>> rot_consistency_mask_(
            new GpuMat<uint8_t>(cost_map_->GetWidth(), cost_map_->GetHeight(),
                                cost_map_->GetDepth()));
        consistency_mask_->Rotate(rot_consistency_mask_.get());
        consistency_mask_.swap(rot_consistency_mask_);
      }

      sweep_timer.Print(" Sweep " + std::to_string(sweep + 1));
    }

    iter_timer.Print("Iteration " + std::to_string(iter + 1));
  }

  total_timer.Print("Total");
}

void PatchMatchCuda::ComputeCudaConfig() {
  sweep_block_size_.x = THREADS_PER_BLOCK;
  sweep_block_size_.y = 1;
  sweep_block_size_.z = 1;
  sweep_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  sweep_grid_size_.y = 1;
  sweep_grid_size_.z = 1;

  elem_wise_block_size_.x = THREADS_PER_BLOCK;
  elem_wise_block_size_.y = THREADS_PER_BLOCK;
  elem_wise_block_size_.z = 1;
  elem_wise_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.y =
      (depth_map_->GetHeight() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.z = 1;
}

void PatchMatchCuda::InitRefImage() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  ref_width_ = ref_image.GetWidth();
  ref_height_ = ref_image.GetHeight();

  // Upload to device.
  ref_image_.reset(new GpuMatRefImage(ref_width_, ref_height_));
  const std::vector<uint8_t> ref_image_array =
      ref_image.GetBitmap().ConvertToRowMajorArray();
  ref_image_->Filter(ref_image_array.data(), options_.window_radius,
                     options_.window_step, options_.sigma_spatial,
                     options_.sigma_color);

  ref_image_device_.reset(
      new CudaArrayWrapper<uint8_t>(ref_width_, ref_height_, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);

  // Create texture.
  ref_image_texture.addressMode[0] = hipAddressModeBorder;
  ref_image_texture.addressMode[1] = hipAddressModeBorder;
  ref_image_texture.addressMode[2] = hipAddressModeBorder;
  ref_image_texture.filterMode = hipFilterModePoint;
  ref_image_texture.normalized = false;
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));
}

void PatchMatchCuda::InitSourceImages() {
  // Determine maximum image size.
  size_t max_width = 0;
  size_t max_height = 0;
  for (const auto image_idx : problem_.src_image_idxs) {
    const Image& image = problem_.images->at(image_idx);
    if (image.GetWidth() > max_width) {
      max_width = image.GetWidth();
    }
    if (image.GetHeight() > max_height) {
      max_height = image.GetHeight();
    }
  }

  // Upload source images to device.
  {
    // Copy source images to contiguous memory block.
    const uint8_t kDefaultValue = 0;
    std::vector<uint8_t> src_images_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const Image& image = problem_.images->at(problem_.src_image_idxs[i]);
      const Bitmap& bitmap = image.GetBitmap();
      uint8_t* dest = src_images_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < image.GetHeight(); ++r) {
        memcpy(dest, bitmap.GetScanline(r), image.GetWidth() * sizeof(uint8_t));
        dest += max_width;
      }
    }

    // Upload to device.
    src_images_device_.reset(new CudaArrayWrapper<uint8_t>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_images_device_->CopyToDevice(src_images_host_data.data());

    // Create source images texture.
    src_images_texture.addressMode[0] = hipAddressModeBorder;
    src_images_texture.addressMode[1] = hipAddressModeBorder;
    src_images_texture.addressMode[2] = hipAddressModeBorder;
    src_images_texture.filterMode = hipFilterModeLinear;
    src_images_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_images_texture,
                                          src_images_device_->GetPtr()));
  }

  // Upload source depth maps to device.
  if (options_.geom_consistency) {
    // change default value to an absurd one
    const float kDefaultValue = -1e20f;
    std::vector<float> src_depth_maps_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const DepthMap& depth_map =
          problem_.depth_maps->at(problem_.src_image_idxs[i]);
      float* dest =
          src_depth_maps_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < depth_map.GetHeight(); ++r) {
        memcpy(dest, depth_map.GetPtr() + r * depth_map.GetWidth(),
               depth_map.GetWidth() * sizeof(float));
        dest += max_width;
      }
    }

    src_depth_maps_device_.reset(new CudaArrayWrapper<float>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_depth_maps_device_->CopyToDevice(src_depth_maps_host_data.data());

    // Create source depth maps texture.
    src_depth_maps_texture.addressMode[0] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[1] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[2] = hipAddressModeBorder;
    // TODO: Check if linear interpolation improves results or not.
    src_depth_maps_texture.filterMode = hipFilterModePoint;
    src_depth_maps_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_depth_maps_texture,
                                          src_depth_maps_device_->GetPtr()));
  }
}

void PatchMatchCuda::InitTransforms() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  //////////////////////////////////////////////////////////////////////////////
  // Generate rotated versions (counter-clockwise) of calibration matrix.
  //////////////////////////////////////////////////////////////////////////////

  for (int i = 0; i < 4; ++i) {
    float K_full_tmp[9];
    float inv_K_full_tmp[9];
    ref_image.Rotate90Multi(i, K_full_tmp, inv_K_full_tmp, ref_R_host_[i], ref_T_host_[i], ref_P_host_[i], ref_inv_P_host_[i], ref_C_host_);
    ref_K_host_[i][0] = K_full_tmp[0];
    ref_K_host_[i][1] = K_full_tmp[1];
    ref_K_host_[i][2] = K_full_tmp[2];
    ref_K_host_[i][3] = K_full_tmp[3];
    ref_K_host_[i][4] = K_full_tmp[4];
    ref_K_host_[i][5] = K_full_tmp[5];

    ref_inv_K_host_[i][0] = inv_K_full_tmp[0];
    ref_inv_K_host_[i][1] = inv_K_full_tmp[1];
    ref_inv_K_host_[i][2] = inv_K_full_tmp[2];
    ref_inv_K_host_[i][3] = inv_K_full_tmp[3];
    ref_inv_K_host_[i][4] = inv_K_full_tmp[4];
    ref_inv_K_host_[i][5] = inv_K_full_tmp[5];
  }

  //max_dist_per_pixel = max_dist_per_pixel_host_;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(max_dist_per_pixel), &max_dist_per_pixel_host_, sizeof(float), 0,
                                    hipMemcpyHostToDevice));
  // copy
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_C), ref_C_host_, sizeof(float) * 3, 0, hipMemcpyHostToDevice));

  // Bind 0 degrees version to constant global memory.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[0], sizeof(float) * 6, 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[0], sizeof(float) * 6, 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_R), ref_R_host_[0], sizeof(float) * 9, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_T), ref_T_host_[0],
                                    sizeof(float) * 3, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_P), ref_P_host_[0],
                                    sizeof(float) * 16, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_P), ref_inv_P_host_[0],
                                    sizeof(float) * 16, 0,
                                    hipMemcpyHostToDevice));

  //////////////////////////////////////////////////////////////////////////////
  // Upload P, inv_P, C for source images
  //////////////////////////////////////////////////////////////////////////////
  const size_t kNumTformParams = 16 + 16 + 3;
  float poses_host_data[kNumTformParams * problem_.src_image_idxs.size()];

  int offset = 0;
  for (const auto image_idx : problem_.src_image_idxs) {
    const Image &image = problem_.images->at(image_idx);

    float K_full[9];
    float inv_K_full[9];
    float R[9];
    float T[3];
    float P[16];
    float inv_P[16];
    float C[3];

    // because the point is in scene coorindaste frame, hence we should not rotate source images
    image.Original(K_full, inv_K_full, R, T, P, inv_P, C);

    memcpy(poses_host_data + offset, P, 16 * sizeof(float));
    offset += 16;
    memcpy(poses_host_data + offset, inv_P, 16 * sizeof(float));
    offset += 16;
    memcpy(poses_host_data + offset, C, 3 * sizeof(float));
    offset += 3;
  }

  poses_device_.reset(new CudaArrayWrapper<float>(
      kNumTformParams, problem_.src_image_idxs.size(), 1));
  poses_device_->CopyToDevice(poses_host_data);

  poses_texture.addressMode[0] = hipAddressModeBorder;
  poses_texture.addressMode[1] = hipAddressModeBorder;
  poses_texture.addressMode[2] = hipAddressModeBorder;
  poses_texture.filterMode = hipFilterModePoint;
  poses_texture.normalized = false;
  CUDA_SAFE_CALL(hipBindTextureToArray(poses_texture, poses_device_->GetPtr()));
}

void PatchMatchCuda::InitWorkspaceMemory() {
  rand_state_map_.reset(new GpuMatPRNG(ref_width_, ref_height_));

  depth_map_.reset(new GpuMat<float>(ref_width_, ref_height_));
  if (options_.geom_consistency) {
    const DepthMap& init_depth_map =
        problem_.depth_maps->at(problem_.ref_image_idx);
    depth_map_->CopyToDevice(init_depth_map.GetPtr(),
                             init_depth_map.GetWidth() * sizeof(float));
  } else {
    depth_map_->FillWithRandomNumbers(options_.depth_min, options_.depth_max,
                                      *rand_state_map_);
  }

  normal_map_.reset(new GpuMat<float>(ref_width_, ref_height_, 3));

  // Note that it is not necessary to keep the selection probability map in
  // memory for all pixels. Theoretically, it is possible to incorporate
  // the temporary selection probabilities in the global_workspace_.
  // However, it is useful to keep the probabilities for the entire image
  // in memory, so that it can be exported.
  sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                        problem_.src_image_idxs.size()));
  prev_sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                             problem_.src_image_idxs.size()));
  prev_sel_prob_map_->FillWithScalar(0.5f);

  cost_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                    problem_.src_image_idxs.size()));

  const int ref_max_dim = std::max(ref_width_, ref_height_);
  global_workspace_.reset(
      new GpuMat<float>(ref_max_dim, problem_.src_image_idxs.size(), 2));

  consistency_mask_.reset(new GpuMat<uint8_t>(0, 0, 0));

  ComputeCudaConfig();

  if (options_.geom_consistency) {
    const NormalMap& init_normal_map =
        problem_.normal_maps->at(problem_.ref_image_idx);
    normal_map_->CopyToDevice(init_normal_map.GetPtr(),
                              init_normal_map.GetWidth() * sizeof(float));
  } else {
    InitNormalMap<<<elem_wise_grid_size_, elem_wise_block_size_>>>(
        *normal_map_, *rand_state_map_);
  }
}

void PatchMatchCuda::Rotate() {
  rotation_in_half_pi_ = (rotation_in_half_pi_ + 1) % 4;

  size_t width;
  size_t height;
  if (rotation_in_half_pi_ % 2 == 0) {
    width = ref_width_;
    height = ref_height_;
  } else {
    width = ref_height_;
    height = ref_width_;
  }

  // Rotate random map.
  {
    std::unique_ptr<GpuMatPRNG> rotated_rand_state_map(
        new GpuMatPRNG(width, height));
    rand_state_map_->Rotate(rotated_rand_state_map.get());
    rand_state_map_.swap(rotated_rand_state_map);
  }

  // Rotate depth map.
  {
    std::unique_ptr<GpuMat<float>> rotated_depth_map(
        new GpuMat<float>(width, height));
    depth_map_->Rotate(rotated_depth_map.get());
    depth_map_.swap(rotated_depth_map);
  }

  // Rotate normal map.
  {
    std::unique_ptr<GpuMat<float>> rotated_normal_map(
        new GpuMat<float>(width, height, 3));
    normal_map_->Rotate(rotated_normal_map.get());
    normal_map_.swap(rotated_normal_map);
  }

  // Rotate reference image.
  {
    std::unique_ptr<GpuMatRefImage> rotated_ref_image(
        new GpuMatRefImage(width, height));
    ref_image_->image->Rotate(rotated_ref_image->image.get());
    ref_image_->sum_image->Rotate(rotated_ref_image->sum_image.get());
    ref_image_->squared_sum_image->Rotate(
        rotated_ref_image->squared_sum_image.get());
    ref_image_.swap(rotated_ref_image);
  }

  // Bind rotated reference image to texture.
  ref_image_device_.reset(new CudaArrayWrapper<uint8_t>(width, height, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);
  CUDA_SAFE_CALL(hipUnbindTexture(ref_image_texture));
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));

  // Rotate selection probability map.
  prev_sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
  sel_prob_map_->Rotate(prev_sel_prob_map_.get());
  sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));

  // Rotate cost map.
  {
    std::unique_ptr<GpuMat<float>> rotated_cost_map(
        new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
    cost_map_->Rotate(rotated_cost_map.get());
    cost_map_.swap(rotated_cost_map);
  }

  // Rotate calibration.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[rotation_in_half_pi_],
                                    sizeof(float) * 6, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[rotation_in_half_pi_],
                         sizeof(float) * 6, 0, hipMemcpyHostToDevice));

  // Rotate extrinsics
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_R), ref_R_host_[rotation_in_half_pi_],
                         sizeof(float) * 9, 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_T), ref_T_host_[rotation_in_half_pi_],
                         sizeof(float) * 3, 0, hipMemcpyHostToDevice));

  // Rotate Projection Matrix
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_P), ref_P_host_[rotation_in_half_pi_],
                         sizeof(float) * 16, 0,
                         hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_P), ref_inv_P_host_[rotation_in_half_pi_],
                         sizeof(float) * 16, 0,
                         hipMemcpyHostToDevice));

  // Recompute Cuda configuration for rotated reference image.
  ComputeCudaConfig();
}

}  // namespace mvs
}  // namespace colmap
